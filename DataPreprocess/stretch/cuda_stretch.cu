#include "hip/hip_runtime.h"
#include <stdio.h>  
#include <time.h>
#include "export.h"
#include <hip/hip_runtime.h>
#include <cutil.h>

#define FUNCTION 1 // set to 0 for a DLM procedure
#define NN 16 // block dimension

/*  DLM for image contrast stretching on CUDA, called from IDL as   
	  CUDA_STRETCH, LUT, IMIN, IMOUT  (procedure)
	  IMOUT = CUDA_STRETCH(LUT, IMIN) (function)
    where 
	  LUT is a byte lookup table, 
      IMIN is an image band (byte format),
      IMOUT is the contrast stetched result.

    Mort Canty (2010) (with thanks to Jia Tse, 
	          University of Nevada, Las Vegas)

The kernel 
*/
	__global__ void cu_stretch(unsigned char *lut, unsigned char *imin, 
		                       unsigned char *imout, int width, int height)
	{
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		int row = blockIdx.y * blockDim.y + threadIdx.y;
		if (row < 0 || col < 0 || row > height -1 || col > width -1 )
			return;
    // contrast stretch
        long idx = row * width + col;
		imout[idx] = lut[ imin[idx] ];	
	}

/*
The host routine
*/
#if FUNCTION
	IDL_VPTR IDL_CDECL cuda_stretch(int argc, IDL_VPTR argv[]) 
#else
	void IDL_CDECL cuda_stretch(int argc, IDL_VPTR argv[]) 
#endif
	{	 
		FILE *fp;
		fp=fopen("d:\\idl\\projects\\development\\stretch\\cuda_stretch.txt", "a");	
		fprintf (fp, "cuda_stretch ---------------------\n");

	// get the input image byte pointers
		unsigned char * img0Ptr = (unsigned char * ) argv[0]->value.arr->data;
		unsigned char * img1Ptr = (unsigned char * ) argv[1]->value.arr->data;

    // get the dimensions of input image band
		long ndim = argv[1]->value.arr->n_dim;
        long * dim = argv[1]->value.arr->dim;
        long cols = dim[0];
		long rows = dim[1];
    // create the output array
        IDL_VPTR ivOutArray;
		unsigned char * imgOutPtr = (unsigned char * ) IDL_MakeTempArray( (int) IDL_TYP_BYTE, ndim, 
			dim, IDL_ARR_INI_ZERO, &ivOutArray);	

		hipEvent_t start,stop;
		float elapsedTime;

        fprintf (fp, "transferring arrays to device ...\ncols = %i rows = %i\n",cols,rows);
		hipEventCreate(&start);
        hipEventCreate(&stop);

		hipEventRecord(start,0);
	// set up the device variables to hold the data from the host
		unsigned char *a0_d;		// Pointer to device array for LUT
		unsigned char *a1_d;		// Pointer to device array for image band
		unsigned char *a2_d;	    // Pointer to device array for image output
		const long N = cols * rows;	// Number of elements in arrays
		size_t size0 = 256;
		size_t size1 = N * sizeof(unsigned char);
		hipMalloc((void **) &a0_d, size0);   // Allocate LUT array on device
		hipMalloc((void **) &a1_d, size1);   // Allocate image band array on device
		hipMalloc((void **) &a2_d, size1);   // Allocate output array on device

		hipMemcpy(a0_d, img0Ptr, size0, hipMemcpyHostToDevice);
		hipMemcpy(a1_d, img1Ptr, size1, hipMemcpyHostToDevice);

		hipEventRecord(stop,0);
		hipEventElapsedTime(&elapsedTime,start,stop);
		fprintf (fp, "time required   %3.1f ms\n", elapsedTime );

	// set up device configurations
		dim3 block(NN,NN);		
		dim3 grid (cols/NN +(cols%NN == 0 ? 0:1), rows/NN + (rows%NN == 0 ? 0:1));	

        fprintf (fp, "launching kernel ...\n");	     
    // launch the kernel
        hipEventRecord(start,0);	
		cu_stretch <<< grid, block >>> (a0_d, a1_d, a2_d, cols, rows);
	// synchronize the threads 
		hipDeviceSynchronize();
		hipEventRecord(stop,0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsedTime,start,stop);
		fprintf (fp, "time required   %3.1f ms\n", elapsedTime );

        fprintf (fp, "transferring result to host ...\n");	
    // return result to the host
		hipEventRecord(start,0);
		hipMemcpy(imgOutPtr, a2_d, size1, hipMemcpyDeviceToHost);
		hipEventRecord(stop,0);
		hipEventElapsedTime(&elapsedTime,start,stop);
		fprintf (fp, "time required   %3.1f ms\n", elapsedTime );
    // clean up
		hipEventDestroy(start);
		hipEventDestroy(stop);
		hipFree(a0_d);
		hipFree(a1_d);
		hipFree(a2_d);
		fclose(fp);

#if FUNCTION 
        return ivOutArray;  
#else
		IDL_VarCopy(ivOutArray,argv[2]);
#endif
	}
	
// the entry point, which loads the routine into IDL 
        int IDL_Load(void) 
		{
#if FUNCTION 
        static IDL_SYSFUN_DEF2 function_addr[] = { 
        { (IDL_SYSRTN_GENERIC) cuda_stretch, "CUDA_STRETCH", 2, 2, 0, 0 } 
		}; 
        return IDL_SysRtnAdd(function_addr, IDL_TRUE, 1);  
#else 
        static IDL_SYSFUN_DEF2 procedure_addr[] = { 
        { (IDL_SYSRTN_GENERIC) cuda_stretch, "CUDA_STRETCH", 3, 3, 0, 0 } 
        }; 
        return IDL_SysRtnAdd(procedure_addr, IDL_FALSE, 1); 
#endif
        }
