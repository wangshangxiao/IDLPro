#include "hip/hip_runtime.h"
#include <stdio.h>  
#include "export.h"
#include <hip/hip_runtime.h>
#include <cutil.h>

/* 
DLM for calculating NDVI on CUDA, called from IDL as   

	  CUDA_NDVI, NIR, RED, NDVI 

where NIR and RED are input spectral bands (byte) and ndvi is output (can be undefined)
*/

// the kernel 
	__global__ void cu_ndvi(unsigned char *a, unsigned char *b, float *out, int width, int height)
	{
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		int row = blockIdx.y * blockDim.y + threadIdx.y;
		if (row < 0 || col < 0 || row > height -1 || col > width -1 )
			return;
		
		long idx = row * width + col;
		if (((float)a[idx] + (float)b[idx]) == 0) 
			out[idx] = 0.0;
		else
			out[idx] = ((float)a[idx] - (float)b[idx])/((float)a[idx] + (float)b[idx]);
	}

// the host (DLM) routine
	void IDL_CDECL cuda_ndvi(int argc, IDL_VPTR argv[]) 
	{	 
	// grab the input image byte pointers
		unsigned char * img0Ptr = (unsigned char * ) argv[0]->value.arr->data;
		unsigned char * img1Ptr = (unsigned char * ) argv[1]->value.arr->data;

    // get the dimensions (same for all three arrays)
		IDL_LONG ndim = argv[0]->value.arr->n_dim;
        IDL_LONG * dim = argv[0]->value.arr->dim;
        IDL_LONG cols = dim[0];
		IDL_LONG rows = dim[1];
    // create the output array
        IDL_VPTR ivOutArray;
		float * imgOutPtr = (float * ) IDL_MakeTempArray( (int) IDL_TYP_FLOAT, ndim, 
			dim, IDL_ARR_INI_ZERO, &ivOutArray);				

	//Setting up the device variables to hold the data from the host
		unsigned char * a0_d;		// Pointer to device array for image 0
		unsigned char * a1_d;		// Pointer to device array for image 1
		float * a2_d;		        // Pointer to device array for image output
		const long N = cols * rows;	// Number of elements in arrays
		size_t size = N * sizeof(unsigned char);
		hipMalloc((void **) &a0_d, size);   // Allocate array on device
		hipMalloc((void **) &a1_d, size);   // Allocate array on device
		hipMalloc((void **) &a2_d, N * sizeof(float));   // Allocate array on device, can be left blank

		hipMemcpy(a0_d, img0Ptr, size, hipMemcpyHostToDevice);
		hipMemcpy(a1_d, img1Ptr, size, hipMemcpyHostToDevice);

	// Setting up device configurations
		dim3 block(16,16);		//16 X 16 blocks for a total of 256 threads
		dim3 grid (cols/16 +(cols%16 == 0 ? 0:1), rows/16 + (rows%16 == 0 ? 0:1));	

    //Actual call to the device for processing.
		cu_ndvi <<< grid, block >>> (a0_d, a1_d, a2_d, cols, rows);
	//Synchronize the threads and stop the timer
		hipDeviceSynchronize();

		hipMemcpy(imgOutPtr, a2_d, sizeof(float)*N, hipMemcpyDeviceToHost);

	//	copy the temporary array to the IDL output parameter
		IDL_VarCopy(ivOutArray,argv[2]);

		hipFree(a0_d);
		hipFree(a1_d);
		hipFree(a2_d);
	}
	
// the entry point, which loads the routine into IDL 
	    int IDL_Load(void) 
    { 
       static IDL_SYSFUN_DEF2 procedure_addr[] = { 
        { (IDL_SYSRTN_GENERIC) cuda_ndvi, "CUDA_NDVI", 0, 3, 0, 0 } 
       }; 
       return IDL_SysRtnAdd(procedure_addr, IDL_FALSE, 1); 
    }  